

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define DataType float

double getTimer() {
  struct timeval tp;
  gettimeofday(&tp,NULL);
  return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

// Compute C = A * B
__global__ void gemm(DataType *A, DataType *B, DataType *C, int numARows,
                     int numAColumns, int numBRows, int numBColumns)
{
  //@@ Insert code to implement matrix multiplication here
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  if ((col >= numBColumns) || (row >= numARows))
    return;

  DataType tmpSum = 0.0;
  for (int k = 0; k < numAColumns; k++)
  {
    tmpSum += A[row * numAColumns + k] * B[k * numBColumns + col];
  }
  C[row * numBColumns + col] = tmpSum;
}

int main(int argc, char **argv)
{

  DataType *hostA;     // The A matrix
  DataType *hostB;     // The B matrix
  DataType *hostC;     // The output C matrix
  DataType *resultRef; // The reference result
  DataType *deviceA;
  DataType *deviceB;
  DataType *deviceC;
  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;
  int numCColumns;

  //@@ Insert code below to read in numARows, numAColumns, numBColumns from args

  numARows = atoi(argv[1]);
  numAColumns = atoi(argv[2]);
  numBRows = atoi(argv[3]);
  numBColumns = atoi(argv[4]);
  numCRows = numARows;
  numCColumns = numBColumns;
  printf("Input matrix dim (%d x %d) (%d x %d) (%d x %d)\n", numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);

  //@@ Insert code below to allocate Host memory for input and output

  int totalSizeA = numARows * numAColumns * sizeof(DataType);
  int totalSizeB = numBRows * numBColumns * sizeof(DataType);
  int totalSizeC = numCRows * numCColumns * sizeof(DataType);
  hostA = (DataType *)malloc(totalSizeA);
  hostB = (DataType *)malloc(totalSizeB);
  hostC = (DataType *)malloc(totalSizeC);
  resultRef = (DataType *)malloc(numCRows * numCColumns * sizeof(DataType));

  //@@ Insert code below to initialize hostA and hostB to random numbers, and create reference result in CPU

  for (int i = 0; i < numARows; i++)
  {
    for (int j = 0; j < numAColumns; j++)
    {
      DataType randomNumber = (DataType)rand() / RAND_MAX;
      hostA[i * numAColumns + j] = randomNumber;
    }
  }

  for (int i = 0; i < numBRows; i++)
  {
    for (int j = 0; j < numBColumns; j++)
    {
      DataType randomNumber = (DataType)rand() / RAND_MAX;
      hostB[i * numBColumns + j] = randomNumber;
    }
  }

  for (int i = 0; i < numARows; i++)
  {
    for (int j = 0; j < numBColumns; j++)
    {
      resultRef[i * numBColumns + j] = 0.0;
      for (int k = 0; k < numAColumns; k++)
      {
        resultRef[i * numBColumns + j] += hostA[i * numAColumns + k] * hostB[k * numBColumns + j];
      }
    }
  }

  //@@ Insert code below to allocate GPU memory here

  hipMalloc(&deviceA, totalSizeA);
  hipMalloc(&deviceB, totalSizeB);
  hipMalloc(&deviceC, totalSizeC);

  //@@ Insert code to below to Copy memory to the GPU here

  double start = getTimer();
  hipMemcpy(deviceA, hostA, totalSizeA, hipMemcpyHostToDevice);
  hipMemcpy(deviceB, hostB, totalSizeB, hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  double duration = getTimer() - start;
  printf("Host to Device Time: %f\n", duration);

  //@@ Initialize the grid and block dimensions here

  int threadPerBlockX = 32;
  int threadPerBlockY = 32;
  int blockNumX = (numCColumns + threadPerBlockX - 1) / threadPerBlockX;
  int blockNumY = (numCRows + threadPerBlockY - 1) / threadPerBlockY;
  printf("threads per block x: %i y: %i\n", threadPerBlockX, threadPerBlockY);
  printf("blocks num x: %i, y: %i \n", blockNumX, blockNumY);

  //@@ Launch the GPU Kernel here

  start = getTimer();
  gemm<<<dim3(blockNumX, blockNumY, 1), dim3(threadPerBlockX, threadPerBlockY, 1)>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns);
  hipDeviceSynchronize();
  duration = getTimer() - start;
  printf("CUDA Kernel: %f\n", duration);

  //@@ Copy the GPU memory back to the CPU here

  start = getTimer();
  hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(DataType), hipMemcpyDeviceToHost);
  hipDeviceSynchronize();
  duration = getTimer() - start;
  printf("Device to Host Time: %f\n", duration);

  //@@ Insert code below to compare the output with the reference

  bool allClose = true;
  for (int i = 0; i < numCRows; ++i)
  {
    for (int j = 0; j < numCColumns; ++j)
    {
      if (fabs(hostC[i * numCColumns + j] - resultRef[i * numCColumns + j]) > 1e-8)
      {
        allClose = false;
        break;
      }
    }
  }

  allClose ? printf("All good!\n") : printf("Something not equal\n");

  //@@ Free the GPU memory here

  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  //@@ Free the CPU memory here
  free(hostA);
  free(hostB);
  free(hostC);
  free(resultRef);

  return 0;
}
