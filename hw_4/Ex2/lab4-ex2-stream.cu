

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

#define DataType double

__global__ void vecAdd(DataType *in1, DataType *in2, DataType *out, int len) {
  //@@ Insert code to implement vector addition here
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < len) {out[idx] = in1[idx] + in2[idx];}
  else {return;}
}

//@@ Insert code to implement timer start

double getTimer() {
  struct timeval tp;
  gettimeofday(&tp,NULL);
  return ((double)tp.tv_sec + (double)tp.tv_usec * 1.e-6);
}

//@@ Insert code to implement timer stop


int main(int argc, char **argv) {
  
  int inputLength;
  int nStreams;
  int segSize;
  // DataType *deviceMemory;
  DataType *deviceInput1;
  DataType *deviceInput2;
  DataType *deviceOutput;
  DataType *hostInput1;
  DataType *hostInput2;
  DataType *hostOutput;
  DataType *resultRef;

  //@@ Insert code below to read in inputLength from args
  
  inputLength = atoi(argv[1]);
  nStreams = atoi(argv[2]);
  segSize = inputLength / nStreams;

  printf("The input length is %d\n", inputLength);
  printf("The segment size is %d\n", segSize);
  printf("The segment num is %d\n", nStreams);

  
  //@@ Insert code below to allocate Host memory for input and output

  int inputActualSize = inputLength * sizeof(DataType);
  hipHostAlloc((void **) &hostInput1, inputActualSize, hipHostMallocDefault);
  hipHostAlloc((void **) &hostInput2, inputActualSize, hipHostMallocDefault);
  hipHostAlloc((void **) &hostOutput, inputActualSize, hipHostMallocDefault);
  resultRef = (DataType*)malloc(inputActualSize);
  
  //@@ Insert code below to initialize hostInput1 and hostInput2 to random numbers, and create reference result in CPU

  for (int i = 0; i < inputLength; i++) {
    DataType randomNumber1 = (DataType) rand() / RAND_MAX;
    DataType randomNumber2 = (DataType) rand() / RAND_MAX;
    hostInput1[i] = randomNumber1;
    hostInput2[i] = randomNumber2;
    resultRef[i] = randomNumber1 + randomNumber2;
  }

  //@@ Insert code below to allocate GPU memory here

  hipMalloc(&deviceInput1, inputActualSize);
  hipMalloc(&deviceInput2, inputActualSize);
  hipMalloc(&deviceOutput, inputActualSize);


  hipStream_t stream[nStreams];

  for(int i = 0; i < nStreams; i++)
    hipStreamCreate(&stream[i]);


  //@@ Initialize the 1D grid and block dimensions here

  int threadPerBlock = 64;
  int blockNum = (segSize + threadPerBlock - 1) / threadPerBlock;
  printf("threads per block: %i \n", threadPerBlock);
  printf("blocks num: %i \n", blockNum);
  

  double start = getTimer();

  for(int i = 0; i < nStreams; i++){
    int offset = i * segSize;
    hipMemcpyAsync(&deviceInput1[offset], &hostInput1[offset], segSize * sizeof(DataType), hipMemcpyHostToDevice, stream[i]);
    hipMemcpyAsync(&deviceInput2[offset], &hostInput2[offset],  segSize * sizeof(DataType), hipMemcpyHostToDevice, stream[i]);
    vecAdd<<<blockNum,threadPerBlock,0,stream[i]>>>(deviceInput1 + offset, deviceInput2 + offset, deviceOutput + offset, segSize);
    hipMemcpyAsync(&hostOutput[offset], &deviceOutput[offset],  segSize * sizeof(DataType), hipMemcpyDeviceToHost, stream[i]);
  }
  
  hipDeviceSynchronize();

  double duration = getTimer() - start;
  printf("Total Time: %f\n", duration);


  for(int i = 0; i < nStreams; i++) {
    hipStreamDestroy(stream[i]);
  }
  //@@ Insert code below to compare the output with the reference

  bool allClose = true;
  for (int i = 0; i < inputLength; i++) {
    if (fabs(hostOutput[i] - resultRef[i]) > 1e-4) {
      allClose = false;
      break;
    }
  }

  allClose ? printf("All good!\n") : printf("Something not equal\n");

  //@@ Free the GPU memory here
  for (int i=0; i < nStreams; i++)
    hipStreamDestroy(stream[i]);
  
  // cudaFree(deviceMemory);
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  
  //@@ Free the CPU memory here
  hipFree(hostInput1);
  hipFree(hostInput2);
  hipFree(hostOutput);
  free(resultRef);

  return 0;
}
